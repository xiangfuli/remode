#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_texture_types.h"

namespace rmd {
    texture<float, hipTextureType2D, hipReadModeElementType> ref_img_tex;
    texture<float, hipTextureType2D, hipReadModeElementType> curr_img_tex;

    texture<float, hipTextureType2D, hipReadModeElementType> mu_tex;
    texture<float, hipTextureType2D, hipReadModeElementType> sigma_tex;
    texture<float, hipTextureType2D, hipReadModeElementType> a_tex;
    texture<float, hipTextureType2D, hipReadModeElementType> b_tex;

    texture<int, hipTextureType2D, hipReadModeElementType> convergence_tex;
    texture<float2, hipTextureType2D, hipReadModeElementType> epipolar_matches_tex;

    texture<float, hipTextureType2D, hipReadModeElementType> g_tex;

    // Pre-computed template statistics
    texture<float, hipTextureType2D, hipReadModeElementType> sum_templ_tex;
    texture<float, hipTextureType2D, hipReadModeElementType> const_templ_denom_tex;
}

#include "depthmap.h"
#include <thread>
#include <chrono>
int main() {
    rmd::Depthmap depth_map(1000000, 200, 400, 400, 500, 100);

    while(1) {
        std::this_thread::sleep_for(std::chrono::seconds(1));
    }

    return 0;
}